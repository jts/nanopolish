#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "gpu_aligner.h"
#include <vector>
#include "nanopolish_profile_hmm_r9.h"

int gpu_aligner_debug = 0;

#define MAX_STATES 256

#define EXPAND_TO_STRING(X) #X
#define TO_STRING(X) EXPAND_TO_STRING(X)
#define CU_CHECK_ERR(X) if (X != hipSuccess){printf("CUDA error: <<%s>> at line %s\n", hipGetErrorString(X), TO_STRING(__LINE__));throw std::runtime_error("CUDA ERROR");}

__device__ float logsumexpf(float x, float y){
    if(x == -INFINITY && y == -INFINITY){
        return -INFINITY;
    }
    float result = fmax(x, y) + log1pf(expf(-fabsf(y - x)));
    return result;
}

__device__ float lp_match_r9(int rank,
                             float mean,
                             float pore_mean,
                             float pore_stdv,
                             float pore_log_level_stdv,
                             float scale,
                             float shift,
                             float var,
                             float logVar){

    float log_inv_sqrt_2pi = logf(0.3989422804014327);

    float level = mean;
    float gaussian_mean = scale * pore_mean + shift;
    float gaussian_stdv = pore_stdv * var;
    float gaussian_log_level_stdv = pore_log_level_stdv + logVar;

    float a = (level - gaussian_mean) / gaussian_stdv;
    float emission = log_inv_sqrt_2pi - gaussian_log_level_stdv + (-0.5f * a * a);
    return emission;

}

__global__ void getScoresMod (float * poreModelDev,
                              int * readLengthsDev,
                              int * eventStartsDev,
                              int * eventStridesDev,
                              float * eventsPerBaseDev,
                              float * scaleDev,
                              float * shiftDev,
                              float * varDev,
                              float * logVarDev,
                              int * eventOffsetsDev,
                              float * eventMeansDev,
                              int * modelOffsetsDev,
                              float * preFlankingDev,
                              float * postFlankingDev,
                              int * sequenceLengthsDev,
                              int * sequenceOffsetsDev,
                              int * kmerRanksDev,
                              int * seqIdxDev,
                              int * readIdxDev,
                              int numScores,
                              float * returnValuesDev){

    bool debug = false;
    if ((threadIdx.x == 0) && (blockIdx.x == 0)){
        debug = false;
    }

    // get buffer indices
    int scoreIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (scoreIdx < numScores) {

        int readIdx = readIdxDev[scoreIdx];
        int seqIdx = seqIdxDev[scoreIdx];

        // get read statistics
        int numEvents = readLengthsDev[readIdx];
        float read_events_per_base = eventsPerBaseDev[readIdx];
        int e_start = eventStartsDev[readIdx]; // Event start for read
        int e_stride = eventStridesDev[readIdx];
        int e_offset = eventOffsetsDev[readIdx]; // Within the event means etc, the offset needed for this block to get a specific event
        int m_offset = modelOffsetsDev[readIdx];
        float scale = scaleDev[readIdx];
        float shift = shiftDev[readIdx];
        float var = varDev[readIdx];
        float logVar = logVarDev[readIdx];

        // get sequence statistics
        int numKmers = sequenceLengthsDev[seqIdx];
        int seqOffset = sequenceOffsetsDev[seqIdx];

        int lastRowIdx = numEvents - 1;
        int lastKmerIdx = numKmers - 1;

        float returnValue = -INFINITY; //Used to sum over the last column.
        float prevProbabilities[MAX_STATES];

        int numBlocks = numKmers + 2;
        int numStates = numBlocks * PSR9_NUM_STATES; // 3 blocks per kmer and then 3 each for start and end state.

        if (debug) {
            printf("Kernel 1 >>> Num Kmers is %i\n", numKmers);
            printf("Kernel 1 >>> n_states %i\n", numStates);
            printf("Kernel 1 >>> num events in read is  %i\n", numEvents);
            printf("Kernel 1 >>> event offset is  %i\n", e_offset);
        }

        // Initialise the prev probabilities vector
        for (int i = 0; i < numStates - PSR9_NUM_STATES; i++) {
            prevProbabilities[i] = -INFINITY;
        }
        for (int i = numStates - PSR9_NUM_STATES; i < numStates; i++) {
            prevProbabilities[i] = 0.0f;
        }

        bool rc = false;
        if (e_stride == -1) {
            rc = true;
        }

        float p_stay = 1 - (1 / read_events_per_base);
        float p_skip = 0.0025;
        float p_bad = 0.001;
        float p_bad_self = p_bad;
        float p_skip_self = 0.3;
        float p_mk = p_skip; // probability of not observing an event at all
        float p_mb = p_bad; // probabilty of observing a bad event
        float p_mm_self = p_stay; // probability of observing additional events from this k-mer
        float p_mm_next = 1.0f - p_mm_self - p_mk - p_mb; // normal movement from state to state
        // transitions from event split state in previous block
        float p_bb = p_bad_self;
        float p_bk, p_bm_next, p_bm_self;
        p_bk = p_bm_next = p_bm_self = (1.0f - p_bb) / 3;
        // transitions from kmer skip state in previous block
        float p_kk = p_skip_self;
        float p_km = 1.0f - p_kk;
        // We assign some transition probabilities. I believe this is correct and they don't vary by location in the sequence
        float lp_mk = logf(p_mk);
        float lp_mb = logf(p_mb);
        float lp_mm_self = logf(p_mm_self);
        float lp_mm_next = logf(p_mm_next);
        float lp_bb = logf(p_bb);
        float lp_bk = logf(p_bk);
        float lp_bm_next = logf(p_bm_next);
        float lp_bm_self = logf(p_bm_self);
        float lp_kk = logf(p_kk);
        float lp_km = logf(p_km);
        float lp_sm, lp_ms;
        lp_sm = lp_ms = 0.0f;

        // the penalty is controlled by the transition probability
        float BAD_EVENT_PENALTY = 0.0f;

        //Fill out the dynamic programming table
        for (int row = 1; row < numEvents + 1; row++) {
            //row-specific values
            int event_idx = e_start + (row - 1) * e_stride;
            float eventMean = eventMeansDev[e_offset + row - 1];
            float preFlank = preFlankingDev[e_offset + row - 1];
            float postFlank = postFlankingDev[e_offset + row - 1];

            float lp_emission_b = BAD_EVENT_PENALTY;

            //Initialise temp registers
            float prevMatch = prevProbabilities[PSR9_MATCH];;
            float prevSkip = prevProbabilities[PSR9_KMER_SKIP];
            float prevBad = prevProbabilities[PSR9_BAD_EVENT];

            for (int blkIdx = 1; blkIdx < numBlocks - 1; blkIdx++) {
                int curBlockIdx = blkIdx;
                int prevBlockIdx = curBlockIdx - 1;
                int prevBlockOffset = PSR9_NUM_STATES * prevBlockIdx;
                int curBlockOffset = PSR9_NUM_STATES * curBlockIdx;

                int kmerIdx = blkIdx - 1; // because there is a start block with no associated kmer
                uint32_t rank = kmerRanksDev[seqOffset + kmerIdx + (numKmers *
                                                                    rc)];

                float pore_mean = poreModelDev[m_offset + rank * 3];
                float pore_stdv = poreModelDev[m_offset + rank * 3 + 1];
                float pore_log_level_stdv = poreModelDev[m_offset + rank * 3 + 2];

                float lp_emission_m = lp_match_r9(rank,
                                                  eventMean,
                                                  pore_mean,
                                                  pore_stdv,
                                                  pore_log_level_stdv,
                                                  scale,
                                                  shift,
                                                  var,
                                                  logVar);

                // Get all the scores for a match
                float curMatch = prevProbabilities[curBlockOffset + PSR9_MATCH];
                float curBad = prevProbabilities[curBlockOffset + PSR9_BAD_EVENT];
                float curSkip = prevProbabilities[curBlockOffset + PSR9_KMER_SKIP];

                float HMT_FROM_SAME_M = lp_mm_self + curMatch;
                float HMT_FROM_PREV_M = lp_mm_next + prevMatch;
                float HMT_FROM_SAME_B = lp_bm_self + curBad;
                float HMT_FROM_PREV_B = lp_bm_next + prevBad;
                float HMT_FROM_PREV_K = lp_km + prevSkip;

                // m_s is the probability of going from the start state
                // to this kmer. The start state is (currently) only
                // allowed to go to the first kmer. If ALLOW_PRE_CLIP
                // is defined, we allow all events before this one to be skipped,
                // with a penalty;
                float HMT_FROM_SOFT = (kmerIdx == 0 &&
                                       (event_idx == e_start ||
                                        (HAF_ALLOW_PRE_CLIP))) ? lp_sm + preFlank : -INFINITY;

                // calculate the score
                float sum = HMT_FROM_SAME_M;
                sum = logsumexpf(sum, HMT_FROM_SOFT);
                sum = logsumexpf(sum, HMT_FROM_PREV_M);
                sum = logsumexpf(sum, HMT_FROM_SAME_B);
                sum = logsumexpf(sum, HMT_FROM_PREV_B);
                sum = logsumexpf(sum, HMT_FROM_PREV_K);
                sum += lp_emission_m;

                float newMatchScore = sum;

                // Calculate the bad event scores
                // state PSR9_BAD_EVENT
                HMT_FROM_SAME_M = lp_mb + curMatch;
                HMT_FROM_PREV_M = -INFINITY;
                HMT_FROM_SAME_B = lp_bb + prevBad;
                HMT_FROM_PREV_B = -INFINITY;
                HMT_FROM_PREV_K = -INFINITY;
                HMT_FROM_SOFT = -INFINITY;

                sum = HMT_FROM_SAME_M;
                sum = logsumexpf(sum, HMT_FROM_SAME_B);
                sum += lp_emission_b;

                float newBadEventScore = sum;

                // Write row out. prevProbabilities now becomes "current probabilities" for evaluating skips.
                prevProbabilities[curBlockOffset + PSR9_MATCH] = newMatchScore;
                prevProbabilities[curBlockOffset + PSR9_BAD_EVENT] = newBadEventScore;

                //Update tmp vars
                prevMatch = curMatch;
                prevSkip = curSkip;
                prevBad = prevBad;

                //Now do the non-skip-skip transition. This relies on the updated vector values.
                // state PSR9_KMER_SKIP
                HMT_FROM_PREV_M = lp_mk + prevProbabilities[prevBlockOffset + PSR9_MATCH];
                HMT_FROM_PREV_B = lp_bk + prevProbabilities[prevBlockOffset + PSR9_BAD_EVENT];
                HMT_FROM_PREV_K = lp_kk + prevProbabilities[prevBlockOffset + PSR9_KMER_SKIP];

                sum = HMT_FROM_PREV_M;
                sum = logsumexpf(sum, HMT_FROM_PREV_B);
                sum = logsumexpf(sum, HMT_FROM_PREV_K);
                sum = logsumexpf(sum, HMT_FROM_PREV_M);

                float newSkipScore = sum;

                prevProbabilities[curBlockOffset + PSR9_KMER_SKIP] = newSkipScore;

                //post-clip transition
                if (kmerIdx == lastKmerIdx && ((HAF_ALLOW_POST_CLIP) || row == lastRowIdx)) {
                    float lp1 = lp_ms + prevProbabilities[curBlockOffset + PSR9_MATCH] + postFlank;
                    float lp2 = lp_ms + prevProbabilities[curBlockOffset + PSR9_BAD_EVENT] + postFlank;
                    float lp3 = lp_ms + prevProbabilities[curBlockOffset + PSR9_KMER_SKIP] + postFlank;

                    float end = returnValue;
                    end = logsumexpf(end, lp1);
                    end = logsumexpf(end, lp2);
                    end = logsumexpf(end, lp3);
                    returnValue = end;
                }
            }
        }
        returnValuesDev[scoreIdx] = returnValue;
    }
}


GpuAligner::GpuAligner()
{
    size_t max_reads_per_worker = LOCI_PER_WORKER * MAX_COVERAGE * MAX_NUM_VARIANTS_PER_LOCUS;
    int readsSizeBuffer = max_reads_per_worker * sizeof(int);
    int maxBuffer = max_reads_per_worker * MAX_SEQUENCE_LENGTH * sizeof(int);

    //OLD
    int max_num_sequences = 1;
    int max_sequence_length = 100;

    poreModelInitialized = false;

    CU_CHECK_ERR(hipMalloc((void**)&scaleDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&scaleHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&shiftDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&shiftHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&varDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&varHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&logVarDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&logVarHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&readLengthsDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&readLengthsHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&eventsPerBaseDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&eventsPerBaseHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&eventStartsDev, readsSizeBuffer));
    CU_CHECK_ERR(hipHostAlloc(&eventStartsHost, readsSizeBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&eventStridesDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&eventStridesHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&eventOffsetsDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&eventOffsetsHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&eventMeansDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&eventMeans, maxBuffer , hipHostMallocDefault));
    
    CU_CHECK_ERR(hipMalloc((void**)&modelOffsetsDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&modelOffsetsHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&preFlankingDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&preFlankingHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&postFlankingDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&postFlankingHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&sequenceOffsetsDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&sequenceOffsetsHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&sequenceLengthsDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&sequenceLengthsHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&scoresDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&returnValuesHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&seqIdxDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&seqIdxHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipMalloc((void**)&readIdxDev, maxBuffer));
    CU_CHECK_ERR(hipHostAlloc(&readIdxHost, maxBuffer, hipHostMallocDefault));

    CU_CHECK_ERR(hipHostAlloc(&kmerRanks, maxBuffer , hipHostMallocDefault));
    CU_CHECK_ERR(hipMalloc((void**)&kmerRanksDev, maxBuffer ));

    //
    // Allocate Device memory for pore model
    // 

    // Count the total number of k-mer states across all pore models
    int numModelElements = 0;
    int numModels = 0;
    for(const PoreModel* model : PoreModelSet::get_all_models()) {
        numModelElements += model->states.size();
        numModels += 1;
    }
    //fprintf(stderr, "Initialized %d states from %d models\n", numModelElements, numModels);
    int poreModelEntriesPerState = 3;
    int totalModelEntries = numModelElements * poreModelEntriesPerState;
    CU_CHECK_ERR(hipMalloc((void**)&poreModelDev, totalModelEntries * sizeof(float)));
    CU_CHECK_ERR(hipHostAlloc(&poreModelHost, totalModelEntries * sizeof(float), hipHostMallocDefault));

    //
    // Initialize pore model
    //
    int modelOffset = 0;
    for(const PoreModel* model : PoreModelSet::get_all_models()) {
        modelToOffsetMap[model] = modelOffset;
        fprintf(stderr, "inserted model %s at offset %d\n", PoreModelSet::get_model_key(*model).c_str(), modelOffset);

        int num_states = model->states.size();
        for(int st=0; st<num_states; st++) {
            auto params = model->states[st];
            poreModelHost[modelOffset++] = params.level_mean;
            poreModelHost[modelOffset++] = params.level_stdv;
            poreModelHost[modelOffset++] = params.level_log_stdv;
        }
    }

    fprintf(stderr, "Initialized %d/%d states from %d models\n", modelOffset, numModelElements, numModels);
    assert(modelOffset == totalModelEntries);

    // Allocate host memory for model
    returnValuesHostResultsPointers.resize(max_num_sequences);
    kmerRanksDevPointers.resize(max_num_sequences);
    returnValuesDevResultsPointers.resize(max_num_sequences);

    for (int i =0; i<max_num_sequences;i++){
        hipStreamCreate(&streams[i]);
    }

    // copy over the pore model to the device
    // TODO: move this somewhere else?
    CU_CHECK_ERR(hipMemcpyAsync(poreModelDev, poreModelHost,
                                 totalModelEntries * sizeof(float), hipMemcpyHostToDevice, streams[0]));
}

//Destructor
GpuAligner::~GpuAligner() {
    CU_CHECK_ERR(hipFree(scaleDev));
    CU_CHECK_ERR(hipFree(shiftDev));
    CU_CHECK_ERR(hipFree(varDev));
    CU_CHECK_ERR(hipFree(logVarDev));
    CU_CHECK_ERR(hipFree(eventsPerBaseDev));
    CU_CHECK_ERR(hipFree(readLengthsDev));
    CU_CHECK_ERR(hipFree(eventMeansDev));
    CU_CHECK_ERR(hipFree(eventStartsDev));
    CU_CHECK_ERR(hipFree(eventStridesDev));
    CU_CHECK_ERR(hipFree(eventOffsetsDev));
    CU_CHECK_ERR(hipFree(modelOffsetsDev));
    CU_CHECK_ERR(hipFree(preFlankingDev));
    CU_CHECK_ERR(hipFree(postFlankingDev));
    CU_CHECK_ERR(hipFree(kmerRanksDev));
    CU_CHECK_ERR(hipFree(poreModelDev));
    CU_CHECK_ERR(hipFree(sequenceOffsetsDev));
    CU_CHECK_ERR(hipFree(sequenceLengthsDev));
    CU_CHECK_ERR(hipFree(scoresDev));
    CU_CHECK_ERR(hipFree(seqIdxDev));
    CU_CHECK_ERR(hipFree(readIdxDev));

    CU_CHECK_ERR(hipHostFree(eventMeans));
    CU_CHECK_ERR(hipHostFree(poreModelHost));
    CU_CHECK_ERR(hipHostFree(preFlankingHost));
    CU_CHECK_ERR(hipHostFree(postFlankingHost));
    CU_CHECK_ERR(hipHostFree(kmerRanks));
    CU_CHECK_ERR(hipHostFree(sequenceOffsetsHost));
    CU_CHECK_ERR(hipHostFree(returnValuesHost));
    CU_CHECK_ERR(hipHostFree(readLengthsHost));
    CU_CHECK_ERR(hipHostFree(sequenceLengthsHost));
    CU_CHECK_ERR(hipHostFree(seqIdxHost));
    CU_CHECK_ERR(hipHostFree(readIdxHost));
    CU_CHECK_ERR(hipHostFree(modelOffsetsHost));

    int max_num_sequences = 1;
    for (int i =0; i<max_num_sequences; i++) {
        CU_CHECK_ERR(hipStreamDestroy(streams[i]));
    }
}

std::vector<std::vector<std::vector<double>>> GpuAligner::scoreKernelMod(std::vector<ScoreSet> &scoreSets,
                                                                         uint32_t alignment_flags){

    int numEventsTotal = 0; // The number of events across all scoreSets
    int numSequences = 0; // The number of sequences across all scoreSets
    int kmerOffset = 0;
    int numReads = 0; // The number of reads across all scoreSets
    int numScoreSets = scoreSets.size();

    int rawReadOffset = 0;
    int globalReadIdx = 0;
    int globalSequenceIdx = 0;
    int globalScoreIdx = 0;

    //Loop over every scoreset, filling out buffers and counters
    for (int scoreSetIdx=0; scoreSetIdx < numScoreSets; scoreSetIdx++) {
        auto scoreSet = scoreSets[scoreSetIdx];
        int firstReadIdxinScoreSet = globalReadIdx;

        //Read data
        for (int eventSequenceIdx=0; eventSequenceIdx < scoreSet.rawData.size(); eventSequenceIdx++) {
            auto e = scoreSet.rawData[eventSequenceIdx];
            numReads++;

            //Read statistics - populate host buffers
            scaleHost[globalReadIdx] = e.read->scalings[e.strand].scale;
            shiftHost[globalReadIdx] = e.read->scalings[e.strand].shift;
            varHost[globalReadIdx] = e.read->scalings[e.strand].var;
            logVarHost[globalReadIdx] = e.read->scalings[e.strand].log_var;

            int e_start = e.event_start_idx;
            eventStartsHost[globalReadIdx] = e_start;

            int e_stride = e.event_stride;
            eventStridesHost[globalReadIdx] = e_stride;

            uint32_t e_end = e.event_stop_idx;
            uint32_t n_events;
            if(e_end > e_start)
                n_events = e_end - e_start + 1;
            else
                n_events = e_start - e_end + 1;
            readLengthsHost[globalReadIdx] = n_events;
            numEventsTotal += n_events;

            eventOffsetsHost[globalReadIdx] = rawReadOffset;

            float readEventsPerBase = e.read->events_per_base[e.strand];
            eventsPerBaseHost[globalReadIdx] = readEventsPerBase;

            std::vector<float> pre_flank = make_pre_flanking(e, e_start, n_events);
            std::vector<float> post_flank = make_post_flanking(e, e_start, n_events);

            for (int i=0;i<n_events;i++) {
                auto event_idx =  e_start + i * e_stride;
                auto scaled = e.read->get_drift_scaled_level(event_idx, e.strand); // send the data in drift scaled
                eventMeans[rawReadOffset + i] = scaled;

                //populate the pre/post-flanking data, since it has a 1-1 correspondence with events
                preFlankingHost[rawReadOffset + i] = pre_flank[i];
                postFlankingHost[rawReadOffset + i] = post_flank[i];
            }

            // look up model offset in the map
            const auto& modelOffsetIter = modelToOffsetMap.find(e.pore_model);
            assert(modelOffsetIter != modelToOffsetMap.end());
            modelOffsetsHost[globalReadIdx] = modelOffsetIter->second;

            rawReadOffset += n_events;
            globalReadIdx++;
        }

        auto & sequences = scoreSet.stateSequences;
        numSequences += sequences.size();

        for (int i = 0; i<sequences.size(); i++) {
            auto sequence = sequences[i];

            sequenceOffsetsHost[globalSequenceIdx] = kmerOffset;

            int sequenceLength = sequence.length();
            // TODO: k must be set per read, per score set not fixed
            const uint32_t k = scoreSet.rawData[0].pore_model->k; 
            int numKmers = sequenceLength - k + 1;

            for(size_t ki = 0; ki < numKmers; ++ki) {
                int rank = sequence.get_kmer_rank(ki, k, false);
                kmerRanks[ki + kmerOffset] = rank;
            }

            kmerOffset += numKmers;

            for(size_t ki = 0; ki < numKmers; ++ki) {
                int rank = sequence.get_kmer_rank(ki, k, true);
                kmerRanks[ki + kmerOffset] = rank;
            }

            kmerOffset += numKmers;

            sequenceLengthsHost[globalSequenceIdx] = numKmers;

            // Loop over the raw reads, producing a cartesian product of reads and sequences
            auto numReadsInScoreSet = scoreSet.rawData.size();
            for (int r=0; r<numReadsInScoreSet; r++){
                seqIdxHost[globalScoreIdx] = globalSequenceIdx;
                readIdxHost[globalScoreIdx] = firstReadIdxinScoreSet + r;
                globalScoreIdx++;
            }

            globalSequenceIdx++;
        }
    }

    // All data is now in host buffers - perform memcpys
    //Read statistics
    CU_CHECK_ERR(hipMemcpyAsync(eventStartsDev, eventStartsHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(eventsPerBaseDev, eventsPerBaseHost,
                                 numReads * sizeof(float), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(scaleDev, scaleHost,
                                 numReads * sizeof(float), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(shiftDev, shiftHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(eventStridesDev, eventStridesHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(varDev, varHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(logVarDev, logVarHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    CU_CHECK_ERR(hipMemcpyAsync(readLengthsDev, readLengthsHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    // Read offsets
    CU_CHECK_ERR(hipMemcpyAsync(eventOffsetsDev, eventOffsetsHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    // Model offsets
    CU_CHECK_ERR(hipMemcpyAsync(modelOffsetsDev, modelOffsetsHost,
                                 numReads * sizeof(int), hipMemcpyHostToDevice, streams[0]));

    // Reads + Flanks
    CU_CHECK_ERR(hipMemcpyAsync(eventMeansDev, eventMeans, numEventsTotal * sizeof(float), hipMemcpyHostToDevice, streams[0] ));

    CU_CHECK_ERR(hipMemcpyAsync(preFlankingDev, preFlankingHost, numEventsTotal * sizeof(float), hipMemcpyHostToDevice, streams[0] ));

    CU_CHECK_ERR(hipMemcpyAsync(postFlankingDev, postFlankingHost, numEventsTotal * sizeof(float), hipMemcpyHostToDevice, streams[0] ));

    // Sequence statistics

    CU_CHECK_ERR(hipMemcpyAsync(sequenceLengthsDev, sequenceLengthsHost, numSequences * sizeof(int), hipMemcpyHostToDevice, streams[0] ));

    // Sequence offsets
    CU_CHECK_ERR(hipMemcpyAsync(sequenceOffsetsDev, sequenceOffsetsHost, numSequences * sizeof(int), hipMemcpyHostToDevice, streams[0] ));

    // Sequences
    CU_CHECK_ERR(hipMemcpyAsync(kmerRanksDev, kmerRanks, kmerOffset * sizeof(int), hipMemcpyHostToDevice, streams[0] ));

    // Job details
    CU_CHECK_ERR(hipMemcpyAsync(seqIdxDev, seqIdxHost, globalScoreIdx * sizeof(int), hipMemcpyHostToDevice, streams[0] ));
    CU_CHECK_ERR(hipMemcpyAsync(readIdxDev, readIdxHost, globalScoreIdx * sizeof(int), hipMemcpyHostToDevice, streams[0] ));

    // Launch Kernels

    int blockSize = 32;
    int numBlocks =  (globalScoreIdx + blockSize - 1 ) / blockSize;
    dim3 dimBlock(blockSize);
    dim3 dimGrid(numBlocks);

    //printf("Launching get scores mod kernel\n");
    getScoresMod <<< dimGrid, dimBlock, 0, streams[0]>>> (poreModelDev,
                                                          readLengthsDev,
                                                          eventStartsDev,
                                                          eventStridesDev,
                                                          eventsPerBaseDev,
                                                          scaleDev,
                                                          shiftDev,
                                                          varDev,
                                                          logVarDev,
                                                          eventOffsetsDev,
                                                          eventMeansDev,
                                                          modelOffsetsDev,
                                                          preFlankingDev,
                                                          postFlankingDev,
                                                          sequenceLengthsDev,
                                                          sequenceOffsetsDev,
                                                          kmerRanksDev,
                                                          seqIdxDev,
                                                          readIdxDev,
                                                          globalScoreIdx,
                                                          scoresDev);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
        printf("Errors during kernel execution: %s\n", hipGetErrorString(err));

    hipMemcpyAsync(returnValuesHost, scoresDev, globalScoreIdx * sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipStreamSynchronize(streams[0]);

    //Unpack results
    int k = 0;
    std::vector<std::vector<std::vector<double>>> result(scoreSets.size());

    for(int scoreSetIdx=0; scoreSetIdx<numScoreSets; scoreSetIdx++){
        auto scoreSet = scoreSets[scoreSetIdx];
        int numSequences = scoreSet.stateSequences.size();
        int numReads = scoreSet.rawData.size();
        for (int seqIdx=0; seqIdx<numSequences; seqIdx++){

            std::vector<double> seqScores(numReads);

            for (int readIdx=0; readIdx<numReads; readIdx++){
                float score = returnValuesHost[k];
                seqScores[readIdx] = score;
                k++;
            }

            result[scoreSetIdx].push_back(seqScores);
        }
    }

    return result;
}


std::vector<Variant> GpuAligner::variantScoresThresholded(std::vector<std::vector<Variant>> input_variants_vector,
                                                          std::vector<Haplotype> base_haplotypes,
                                                          std::vector<std::vector<HMMInputData>> event_sequences_vector,
                                                          uint32_t alignment_flags,
                                                          int screen_score_threshold,
                                                          std::vector<std::string> methylation_types) {
    int numScoreSets = base_haplotypes.size();
    std::vector<ScoreSet> scoreSets;
    scoreSets.resize(numScoreSets);

    if(gpu_aligner_debug){
        fprintf(stderr,"Generating variants:\n");
    }

    for(int scoreSetIdx=0; scoreSetIdx<numScoreSets;scoreSetIdx++){

        if(gpu_aligner_debug){
            fprintf(stderr,"scoreSetIdx=%d\t",scoreSetIdx);
        }

        auto input_variants = input_variants_vector[scoreSetIdx];
        auto base_haplotype = base_haplotypes[scoreSetIdx];
        auto event_sequences = event_sequences_vector[scoreSetIdx];

        if (event_sequences.size() > MAX_COVERAGE) {
            event_sequences.resize(MAX_COVERAGE);
        }

        int numVariants = input_variants.size();

        std::vector<Variant> out_variants = input_variants;
        std::vector<Haplotype> variant_haplotypes(numVariants, base_haplotype);

        //loop over the vector, applying the variants to the haplotypes
        for (int i = 0; i<input_variants.size();i++){
            variant_haplotypes[i].apply_variant(input_variants[i]);
        }

        // Make methylated versions of each input sequence. Once for the base haplotype and once each for each variant
        std::vector<HMMInputSequence> sequences;
        std::vector<HMMInputSequence> base_sequence_vector = generate_methylated_alternatives(base_haplotype.get_sequence(),methylation_types);

#ifdef MULTI_MODEL
        std::vector<size_t> num_models_vector;
        std::vector<size_t> score_offsets_vector;
        size_t offset = 0;
        size_t num_models = base_sequence_vector.size();
        num_models_vector.push_back(num_models);
        score_offsets_vector.push_back(offset);
        if(gpu_aligner_debug){
            fprintf(stderr,"num_models_base=%ld,offset_base=%ld\t",num_models,offset);
        }
        offset += num_models;
        for (auto base_sequence: base_sequence_vector){
             sequences.push_back(base_sequence);
        }
#else
        HMMInputSequence base_sequence = base_sequence_vector[0];
        sequences.push_back(base_sequence);
#endif

        for (auto v: variant_haplotypes){
            auto variant_sequence_vector = generate_methylated_alternatives(v.get_sequence(), methylation_types);
#ifdef MULTI_MODEL
            size_t num_models = variant_sequence_vector.size();
            num_models_vector.push_back(num_models);
            score_offsets_vector.push_back(offset);
            if(gpu_aligner_debug){
                fprintf(stderr,"num_models_var=%ld,offset_var=%ld\t",num_models,offset);
            }
            offset += num_models;
            for (auto variant_sequence: variant_sequence_vector){
                sequences.push_back(variant_sequence);
            }
#else
            auto variant_sequence = variant_sequence_vector[0];
            sequences.push_back(variant_sequence);
#endif
        }

        ScoreSet s = {
            sequences,
            event_sequences
#ifdef MULTI_MODEL
            ,
            num_models_vector,
            score_offsets_vector
#endif
        };

        scoreSets[scoreSetIdx] = s;
        if(gpu_aligner_debug){
            fprintf(stderr,"\n");
        }
    }
    if(gpu_aligner_debug){
        fprintf(stderr,"\n");
    }

    std::vector<Variant> v;
    if (!event_sequences_vector.empty()) {

        if(gpu_aligner_debug){
            fprintf(stderr,"Calling scoreKernelMod\n");
        }
        auto scoresMod = scoreKernelMod(scoreSets, alignment_flags);

        if(gpu_aligner_debug){
            fprintf(stderr,"Unpacking scores\n");
        }
        // results are now ready, need to unpack them
        for (int scoreSetIdx=0; scoreSetIdx<numScoreSets; scoreSetIdx++){
            if(gpu_aligner_debug) {
                fprintf(stderr,"scoreSetIdx=%d\t",scoreSetIdx);\
            }
            std::vector<std::vector<double>> scores = scoresMod[scoreSetIdx]; // scores for this candidate, including all variants and base(zeroth)
        #ifdef MULTI_MODEL
            ScoreSet s = scoreSets[scoreSetIdx];
            int numVariants = s.num_models_vector.size() -1; // subtract one for the base sequence
        #else
            int numVariants = scores.size() - 1; // subtract one for the base sequence
        #endif
            int numScores = scores[0].size();
            for (int variantIndex = 0; variantIndex < numVariants; variantIndex++) { // index 0 is the base scores
                double totalScore = 0.0;
                for (int k = 0; k < numScores; k++) {
                    if (fabs(totalScore) < screen_score_threshold) {
                    #ifdef MULTI_MODEL

                        //compute the base score based on the base sequences
                        size_t num_models = s.num_models_vector[0];
                        double num_model_penalty = log(num_models);
                        double score = scores[0][k] - num_model_penalty;
                        for(size_t seq_idx = 1; seq_idx < num_models; ++seq_idx) {
                            double alt_score = scores[seq_idx][k] - num_model_penalty;
                            score = add_logs(score, alt_score);
                        }
                        double baseScore = score;
                        if (k==0 && variantIndex==0 && gpu_aligner_debug){
                            fprintf(stderr,"num_models_base=%ld,offset_base=%ld\t",num_models,0);
                        }

                        if(variantIndex+1 >= s.num_models_vector.size()){ //a sanity check
                            fprintf(stderr,"\nAn invalid memory access occured\nscoreSetIdx=%d, variantIndex=%d, k=%d, \n",scoreSetIdx,variantIndex,k);
                            assert(0);
                        }

                        //compute the variant score based on the variant sequences
                        num_models = s.num_models_vector[variantIndex+1];
                        size_t score_offset = s.score_offsets_vector[variantIndex+1];
                        num_model_penalty = log(num_models);
                        score = scores[score_offset][k] - num_model_penalty;
                        for(size_t seq_idx = 1; seq_idx < num_models; ++seq_idx) {
                            double alt_score = scores[score_offset + seq_idx][k] - num_model_penalty;
                            score = add_logs(score, alt_score);
                        }
                        double variantScore = score;
                        if (k==0 && gpu_aligner_debug) {
                            fprintf(stderr,"num_models_var=%ld,offset_var=%ld\t",num_models,score_offset);
                        }

                    #else
                        double baseScore = scores[0][k];
                        double variantScore = scores[variantIndex + 1][k];
                    #endif
                        totalScore += (variantScore - baseScore);
                    }
                }
                // get the old variant:
                auto unScoredVariant = input_variants_vector[scoreSetIdx][variantIndex];
                unScoredVariant.quality = totalScore;
                unScoredVariant.info = "";
                v.push_back(unScoredVariant);
            }
            if(gpu_aligner_debug){
                fprintf(stderr,"\n");
            }
        }
        if(gpu_aligner_debug){
            fprintf(stderr,"\n");
        }
    }
    return v;
}
